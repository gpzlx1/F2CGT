#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <torch/script.h>

#include "../common.h"
#include "atomic.h"
#include "cub_function.h"
#include "cuda_ops.h"
#include "hashmap.h"

#define BLOCK_SIZE 128
namespace bifeat {

template <typename IdType>
inline torch::Tensor _GetSubIndptr(torch::Tensor seeds, torch::Tensor indptr,
                                   int64_t num_pick, bool replace) {
  int64_t num_items = seeds.numel();
  torch::Tensor sub_indptr = torch::empty(
      (num_items + 1),
      torch::TensorOptions().dtype(indptr.dtype()).device(torch::kCUDA));
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType *>(sub_indptr.data_ptr<IdType>()));

  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(
      it(0), it(num_items),
      [in = seeds.data_ptr<IdType>(), in_indptr = indptr.data_ptr<IdType>(),
       out = thrust::raw_pointer_cast(item_prefix), replace,
       num_pick] __device__(int i) mutable {
        IdType row = in[i];
        IdType begin = in_indptr[row];
        IdType end = in_indptr[row + 1];
        if (replace) {
          out[i] = (end - begin) == 0 ? 0 : num_pick;
        } else {
          out[i] = MIN(end - begin, num_pick);
        }
      });

  cub_exclusiveSum<IdType>(thrust::raw_pointer_cast(item_prefix),
                           num_items + 1);
  return sub_indptr;
}

template <typename IdType>
inline torch::Tensor _GetSubIndptrWithCaching(torch::Tensor seeds,
                                              torch::Tensor cpu_indptr,
                                              torch::Tensor gpu_indptr,
                                              torch::Tensor orig_nids_hashed,
                                              torch::Tensor gpu_nids_hashed,
                                              int64_t num_pick, bool replace) {
  int64_t num_items = seeds.numel();
  torch::Tensor sub_indptr =
      torch::empty((num_items + 1), gpu_indptr.options());
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType *>(sub_indptr.data_ptr<IdType>()));
  int64_t dir_size = orig_nids_hashed.numel();

  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(it(0), it(num_items),
                   [in_hash_size = dir_size, in = seeds.data_ptr<IdType>(),
                    in_cpu_indptr = cpu_indptr.data_ptr<IdType>(),
                    in_gpu_indptr = gpu_indptr.data_ptr<IdType>(),
                    in_hash_key = orig_nids_hashed.data_ptr<IdType>(),
                    in_hash_value = gpu_nids_hashed.data_ptr<int32_t>(),
                    out = thrust::raw_pointer_cast(item_prefix), replace,
                    num_pick] __device__(int i) mutable {
                     Hashmap<IdType> table(in_hash_key, in_hash_value,
                                           in_hash_size);
                     const int64_t pos = table.SearchForPos(in[i]);
                     IdType begin = 0;
                     IdType end = 0;
                     if (pos != -1) {
                       begin = in_gpu_indptr[in_hash_value[pos]];
                       end = in_gpu_indptr[in_hash_value[pos] + 1];
                     } else {
                       begin = in_cpu_indptr[in[i]];
                       end = in_cpu_indptr[in[i] + 1];
                     }
                     if (replace) {
                       out[i] = (end - begin) == 0 ? 0 : num_pick;
                     } else {
                       out[i] = MIN(end - begin, num_pick);
                     }
                   });

  cub_exclusiveSum<IdType>(thrust::raw_pointer_cast(item_prefix),
                           num_items + 1);
  return sub_indptr;
}

template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType *__restrict__ const in_rows,
    const IdType *__restrict__ const in_ptr,
    const IdType *__restrict__ const in_index,
    const IdType *__restrict__ const out_ptr,
    IdType *__restrict__ const out_rows, IdType *__restrict__ const out_cols) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = in_ptr[row];
    const int64_t deg = in_ptr[row + 1] - in_row_start;
    const int64_t out_row_start = out_ptr[out_row];

    if (deg <= num_picks) {
      // just copy row when there is not enough nodes to sample.
      for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const IdType in_idx = in_row_start + idx;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index[in_idx];
      }
    } else {
      // generate permutation list via reservoir algorithm
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        out_cols[out_row_start + idx] = idx;
      }
      __syncthreads();

      for (int idx = num_picks + threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const int num = hiprand(&rng) % (idx + 1);
        if (num < num_picks) {
          // use max so as to achieve the replacement order the serial
          // algorithm would have
          atomic::AtomicMax(out_cols + out_row_start + num, IdType(idx));
        }
      }
      __syncthreads();

      // copy permutation over
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const IdType perm_idx = out_cols[out_row_start + idx] + in_row_start;
        out_rows[out_row_start + idx] = row;
        out_cols[out_row_start + idx] = in_index[perm_idx];
      }
    }
    out_row += 1;
  }
}

template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformReplaceKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const IdType *__restrict__ const in_rows,
    const IdType *__restrict__ const in_ptr,
    const IdType *__restrict__ const in_index,
    const IdType *__restrict__ const out_ptr,
    IdType *__restrict__ const out_rows, IdType *__restrict__ const out_cols) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t in_row_start = in_ptr[row];
    const int64_t out_row_start = out_ptr[out_row];
    const int64_t deg = in_ptr[row + 1] - in_row_start;

    if (deg > 0) {
      // each thread then blindly copies in rows only if deg > 0.
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const int64_t edge = hiprand(&rng) % deg;
        const int64_t out_idx = out_row_start + idx;
        out_rows[out_idx] = row;
        out_cols[out_idx] = in_index[in_row_start + edge];
      }
    }
    out_row += 1;
  }
}

template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformWithCachingKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const int64_t hash_dir_size, const IdType *__restrict__ const in_rows,
    const IdType *__restrict__ const in_gpu_ptr,
    const IdType *__restrict__ const in_cpu_ptr,
    const IdType *__restrict__ const in_gpu_index,
    const IdType *__restrict__ const in_cpu_index,
    const IdType *__restrict__ const out_ptr,
    IdType *__restrict__ const orig_nids_hashed,
    int32_t *__restrict__ const gpu_nids_hashed,
    IdType *__restrict__ const out_rows, IdType *__restrict__ const out_cols) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  Hashmap<IdType> table(orig_nids_hashed, gpu_nids_hashed, hash_dir_size);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t pos = table.SearchForPos(row);
    int64_t in_row_start = 0;
    int64_t deg = 0;
    if (pos != -1) {
      in_row_start = in_gpu_ptr[gpu_nids_hashed[pos]];
      deg = in_gpu_ptr[gpu_nids_hashed[pos] + 1] - in_row_start;
    } else {
      in_row_start = in_cpu_ptr[row];
      deg = in_cpu_ptr[row + 1] - in_row_start;
    }
    const int64_t out_row_start = out_ptr[out_row];

    if (deg <= num_picks) {
      // just copy row when there is not enough nodes to sample.
      for (int idx = threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const IdType in_idx = in_row_start + idx;
        out_rows[out_row_start + idx] = row;
        if (pos != -1) {
          out_cols[out_row_start + idx] = in_gpu_index[in_idx];
        } else {
          out_cols[out_row_start + idx] = in_cpu_index[in_idx];
        }
      }
    } else {
      // generate permutation list via reservoir algorithm
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        out_cols[out_row_start + idx] = idx;
      }
      __syncthreads();

      for (int idx = num_picks + threadIdx.x; idx < deg; idx += BLOCK_SIZE) {
        const int num = hiprand(&rng) % (idx + 1);
        if (num < num_picks) {
          // use max so as to achieve the replacement order the serial
          // algorithm would have
          atomic::AtomicMax(out_cols + out_row_start + num, IdType(idx));
        }
      }
      __syncthreads();

      // copy permutation over
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const IdType perm_idx = out_cols[out_row_start + idx] + in_row_start;
        out_rows[out_row_start + idx] = row;
        if (pos != -1) {
          out_cols[out_row_start + idx] = in_gpu_index[perm_idx];
        } else {
          out_cols[out_row_start + idx] = in_cpu_index[perm_idx];
        }
      }
    }
    out_row += 1;
  }
}

template <typename IdType, int TILE_SIZE>
__global__ void _CSRRowWiseSampleUniformReplaceWithCachingKernel(
    const uint64_t rand_seed, const int64_t num_picks, const int64_t num_rows,
    const int64_t hash_dir_size, const IdType *__restrict__ const in_rows,
    const IdType *__restrict__ const in_gpu_ptr,
    const IdType *__restrict__ const in_cpu_ptr,
    const IdType *__restrict__ const in_gpu_index,
    const IdType *__restrict__ const in_cpu_index,
    const IdType *__restrict__ const out_ptr,
    IdType *__restrict__ const orig_nids_hashed,
    int32_t *__restrict__ const gpu_nids_hashed,
    IdType *__restrict__ const out_rows, IdType *__restrict__ const out_cols) {
  // we assign one warp per row
  assert(blockDim.x == BLOCK_SIZE);

  int64_t out_row = blockIdx.x * TILE_SIZE;
  const int64_t last_row =
      min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

  hiprandStatePhilox4_32_10_t rng;
  hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.x, 0, &rng);

  Hashmap<IdType> table(orig_nids_hashed, gpu_nids_hashed, hash_dir_size);

  while (out_row < last_row) {
    const int64_t row = in_rows[out_row];
    const int64_t pos = table.SearchForPos(row);
    int64_t in_row_start = 0;
    int64_t deg = 0;
    if (pos != -1) {
      in_row_start = in_gpu_ptr[gpu_nids_hashed[pos]];
      deg = in_gpu_ptr[gpu_nids_hashed[pos] + 1] - in_row_start;
    } else {
      in_row_start = in_cpu_ptr[row];
      deg = in_cpu_ptr[row + 1] - in_row_start;
    }
    const int64_t out_row_start = out_ptr[out_row];

    if (deg > 0) {
      // each thread then blindly copies in rows only if deg > 0.
      for (int idx = threadIdx.x; idx < num_picks; idx += BLOCK_SIZE) {
        const int64_t edge = hiprand(&rng) % deg;
        const int64_t out_idx = out_row_start + idx;
        out_rows[out_idx] = row;
        if (pos != -1) {
          out_cols[out_idx] = in_gpu_index[in_row_start + edge];
        } else {
          out_cols[out_idx] = in_cpu_index[in_row_start + edge];
        }
      }
    }
    out_row += 1;
  }
}

std::tuple<torch::Tensor, torch::Tensor> RowWiseSamplingUniformCUDA(
    torch::Tensor seeds, torch::Tensor indptr, torch::Tensor indices,
    int64_t num_picks, bool replace) {
  CHECK_CUDA(seeds);
  PG_ID_TYPE_SWITCH(indptr.dtype(), IdType, {
    int num_rows = seeds.numel();
    torch::Tensor sub_indptr =
        _GetSubIndptr<IdType>(seeds, indptr, num_picks, replace);
    thrust::device_ptr<IdType> item_prefix(
        static_cast<IdType *>(sub_indptr.data_ptr<IdType>()));
    int nnz = item_prefix[num_rows];

    torch::Tensor coo_row = torch::empty(
        nnz, torch::TensorOptions().dtype(seeds.dtype()).device(torch::kCUDA));
    torch::Tensor coo_col = torch::empty(
        nnz,
        torch::TensorOptions().dtype(indices.dtype()).device(torch::kCUDA));

    const uint64_t random_seed = 7777;

    constexpr int TILE_SIZE = 128 / BLOCK_SIZE;
    if (replace) {
      const dim3 block(BLOCK_SIZE);
      const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
      _CSRRowWiseSampleUniformReplaceKernel<IdType, TILE_SIZE><<<grid, block>>>(
          random_seed, num_picks, num_rows, seeds.data_ptr<IdType>(),
          indptr.data_ptr<IdType>(), indices.data_ptr<IdType>(),
          sub_indptr.data_ptr<IdType>(), coo_row.data_ptr<IdType>(),
          coo_col.data_ptr<IdType>());
    } else {
      const dim3 block(BLOCK_SIZE);
      const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
      _CSRRowWiseSampleUniformKernel<IdType, TILE_SIZE><<<grid, block>>>(
          random_seed, num_picks, num_rows, seeds.data_ptr<IdType>(),
          indptr.data_ptr<IdType>(), indices.data_ptr<IdType>(),
          sub_indptr.data_ptr<IdType>(), coo_row.data_ptr<IdType>(),
          coo_col.data_ptr<IdType>());
    }

    return std::make_tuple(coo_row, coo_col);
  });
  return std::make_tuple(torch::Tensor(), torch::Tensor());
}

std::tuple<torch::Tensor, torch::Tensor> RowWiseSamplingUniformWithCachingCUDA(
    torch::Tensor seeds, torch::Tensor gpu_indptr, torch::Tensor cpu_indptr,
    torch::Tensor gpu_indices, torch::Tensor cpu_indices,
    torch::Tensor orig_nids_hashed, torch::Tensor gpu_nids_hashed,
    int64_t num_picks, bool replace) {
  CHECK_CUDA(seeds);
  CHECK_CUDA(gpu_indptr);
  CHECK_CUDA(gpu_indices);
  CHECK_CUDA(orig_nids_hashed);
  CHECK_CUDA(gpu_nids_hashed);
  PG_ID_TYPE_SWITCH(gpu_indptr.dtype(), IdType, {
    int num_rows = seeds.numel();
    torch::Tensor sub_indptr = _GetSubIndptrWithCaching<IdType>(
        seeds, cpu_indptr, gpu_indptr, orig_nids_hashed, gpu_nids_hashed,
        num_picks, replace);
    thrust::device_ptr<IdType> item_prefix(
        static_cast<IdType *>(sub_indptr.data_ptr<IdType>()));
    int nnz = item_prefix[num_rows];

    torch::Tensor coo_row = torch::empty(
        nnz, torch::TensorOptions().dtype(seeds.dtype()).device(torch::kCUDA));
    torch::Tensor coo_col = torch::empty(nnz, gpu_indices.options());

    const uint64_t random_seed = 7777;
    constexpr int TILE_SIZE = 128 / BLOCK_SIZE;
    int dir_size = orig_nids_hashed.numel();
    if (replace) {
      const dim3 block(BLOCK_SIZE);
      const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
      _CSRRowWiseSampleUniformReplaceWithCachingKernel<IdType, TILE_SIZE>
          <<<grid, block>>>(
              random_seed, num_picks, num_rows, dir_size,
              seeds.data_ptr<IdType>(), gpu_indptr.data_ptr<IdType>(),
              cpu_indptr.data_ptr<IdType>(), gpu_indices.data_ptr<IdType>(),
              cpu_indices.data_ptr<IdType>(), sub_indptr.data_ptr<IdType>(),
              orig_nids_hashed.data_ptr<IdType>(),
              gpu_nids_hashed.data_ptr<int32_t>(), coo_row.data_ptr<IdType>(),
              coo_col.data_ptr<IdType>());
    } else {
      const dim3 block(BLOCK_SIZE);
      const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);
      _CSRRowWiseSampleUniformWithCachingKernel<IdType, TILE_SIZE>
          <<<grid, block>>>(
              random_seed, num_picks, num_rows, dir_size,
              seeds.data_ptr<IdType>(), gpu_indptr.data_ptr<IdType>(),
              cpu_indptr.data_ptr<IdType>(), gpu_indices.data_ptr<IdType>(),
              cpu_indices.data_ptr<IdType>(), sub_indptr.data_ptr<IdType>(),
              orig_nids_hashed.data_ptr<IdType>(),
              gpu_nids_hashed.data_ptr<int32_t>(), coo_row.data_ptr<IdType>(),
              coo_col.data_ptr<IdType>());
    }

    return std::make_tuple(coo_row, coo_col);
  });
  return std::make_tuple(torch::Tensor(), torch::Tensor());
}

}  // namespace bifeat