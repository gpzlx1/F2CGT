#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <torch/script.h>

#include "../common.h"
#include "atomic.h"
#include "cub_function.h"
#include "cuda_ops.h"

namespace bifeat {

template <typename IdType>
struct RelabelHashmap {
  __device__ inline RelabelHashmap(IdType* __restrict__ Kptr,
                                   IdType* __restrict__ Vptr, size_t numel)
      : kptr(Kptr), vptr(Vptr), capacity(numel){};

  __device__ inline void Update(IdType key, IdType value) {
    uint32_t delta = 1;
    uint32_t pos = hash(key);
    IdType prev = atomic::AtomicCAS(&kptr[pos], kEmptyKey, key);

    while (prev != key and prev != kEmptyKey) {
      pos = hash(pos + delta);
      delta += 1;
      prev = atomic::AtomicCAS(&kptr[pos], kEmptyKey, key);
    }

    atomic::AtomicMin(vptr + pos, value);
  }

  __device__ inline IdType SearchForPos(IdType key) {
    uint32_t delta = 1;
    uint32_t pos = hash(key);

    while (true) {
      if (kptr[pos] == key) {
        return pos;
      }
      if (kptr[pos] == kEmptyKey) {
        return -1;
      }
      pos = hash(pos + delta);
      delta += 1;
    }
  }

  __device__ inline IdType SearchForValue(IdType key) {
    uint32_t delta = 1;
    uint32_t pos = hash(key);

    while (true) {
      if (kptr[pos] == key) {
        return vptr[pos];
      };
      if (kptr[pos] == kEmptyKey) {
        return -1;
      }
      pos = hash(pos + delta);
      delta += 1;
    }
  }

  __device__ inline uint32_t hash(int32_t key) { return key & (capacity - 1); }

  __device__ inline uint32_t hash(uint32_t key) { return key & (capacity - 1); }

  __device__ inline uint32_t hash(int64_t key) { return key & (capacity - 1); }

  __device__ inline uint32_t hash(uint64_t key) { return key & (capacity - 1); }

  IdType kEmptyKey{-1};
  IdType* kptr;
  IdType* vptr;
  uint32_t capacity{0};
};

inline int _UpPower(int key) {
  int ret = 1 << static_cast<uint32_t>(std::log2(key) + 1);
  return ret;
}

template <typename IdType>
inline std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> Unique(
    torch::Tensor total_tensor) {
  int num_items = total_tensor.numel();
  int dir_size = _UpPower(num_items);

  IdType MAX = std::numeric_limits<IdType>::max();
  torch::Tensor key_tensor = torch::full(
      {
          dir_size,
      },
      -1, total_tensor.options());
  torch::Tensor index_tensor = torch::full(
      {
          dir_size,
      },
      MAX, total_tensor.options());

  // insert
  using it = thrust::counting_iterator<IdType>;
  thrust::for_each(it(0), it(num_items),
                   [key = key_tensor.data_ptr<IdType>(),
                    index = index_tensor.data_ptr<IdType>(),
                    in = total_tensor.data_ptr<IdType>(), num_items,
                    dir_size] __device__(IdType i) mutable {
                     RelabelHashmap<IdType> table(key, index, dir_size);
                     table.Update(in[i], i);
                   });

  // prefix sum
  torch::Tensor item_prefix_tensor =
      torch::empty(num_items + 1, total_tensor.options());
  thrust::device_ptr<IdType> item_prefix(
      static_cast<IdType*>(item_prefix_tensor.data_ptr<IdType>()));
  thrust::for_each(it(0), it(num_items),
                   [key = key_tensor.data_ptr<IdType>(),
                    index = index_tensor.data_ptr<IdType>(),
                    in = total_tensor.data_ptr<IdType>(),
                    count = thrust::raw_pointer_cast(item_prefix), num_items,
                    dir_size] __device__(IdType i) mutable {
                     RelabelHashmap<IdType> table(key, index, dir_size);
                     count[i] = table.SearchForValue(in[i]) == i ? 1 : 0;
                   });
  cub_exclusiveSum<IdType>(thrust::raw_pointer_cast(item_prefix),
                           num_items + 1);

  // unique
  int tot = item_prefix[num_items];
  torch::Tensor unique_tensor = torch::empty(
      {
          tot,
      },
      total_tensor.options());

  torch::Tensor value_tensor = torch::full(
      {
          dir_size,
      },
      -1, total_tensor.options());

  thrust::for_each(it(0), it(num_items),
                   [key = key_tensor.data_ptr<IdType>(),
                    index = index_tensor.data_ptr<IdType>(),
                    in = total_tensor.data_ptr<IdType>(),
                    prefix = thrust::raw_pointer_cast(item_prefix),
                    unique = unique_tensor.data_ptr<IdType>(),
                    cache_value = value_tensor.data_ptr<IdType>(), num_items,
                    dir_size] __device__(IdType i) mutable {
                     RelabelHashmap<IdType> table(key, index, dir_size);
                     IdType pos = table.SearchForPos(in[i]);
                     if (index[pos] == i) {
                       unique[prefix[i]] = in[i];
                       cache_value[pos] = prefix[i];
                     }
                   });

  return {unique_tensor, key_tensor, value_tensor};
}

template <typename IdType>
inline torch::Tensor Relabel(torch::Tensor total_tensor,
                             torch::Tensor key_tensor,
                             torch::Tensor value_tensor) {
  int num_items = total_tensor.numel();
  using it = thrust::counting_iterator<IdType>;
  torch::Tensor relabel_tensor = torch::zeros_like(total_tensor);
  int dir_size = key_tensor.numel();

  thrust::for_each(it(0), it(num_items),
                   [key = key_tensor.data_ptr<IdType>(),
                    value = value_tensor.data_ptr<IdType>(),
                    in = total_tensor.data_ptr<IdType>(),
                    out = relabel_tensor.data_ptr<IdType>(),
                    dir_size] __device__(IdType i) mutable {
                     RelabelHashmap<IdType> table(key, value, dir_size);
                     out[i] = table.SearchForValue(in[i]);
                   });
  return relabel_tensor;
}

std::tuple<torch::Tensor, std::vector<torch::Tensor>> TensorRelabelCUDA(
    std::vector<torch::Tensor> mapping_tensors,
    std::vector<torch::Tensor> requiring_relabel_tensors) {
  std::vector<int64_t> split_sizes;
  for (auto d : requiring_relabel_tensors) {
    split_sizes.push_back(d.numel());
  }

  torch::Tensor total_tensor = torch::cat(requiring_relabel_tensors, 0);

  torch::Tensor unique_tensor, key_tensor, value_tenosr;
  torch::Tensor reindex_tensor;

  PG_ID_TYPE_SWITCH(total_tensor.dtype(), IdType, {
    std::tie(unique_tensor, key_tensor, value_tenosr) =
        Unique<IdType>(torch::cat(mapping_tensors, 0));
    reindex_tensor = Relabel<IdType>(total_tensor, key_tensor, value_tenosr);
  });

  std::vector<torch::Tensor> ret =
      reindex_tensor.split_with_sizes(split_sizes, 0);

  return std::make_tuple(unique_tensor, ret);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor> TensorOrderUniqueCUDA(
    std::vector<torch::Tensor> tensors) {
  torch::Tensor total_tensor = torch::cat(tensors, 0);
  PG_ID_TYPE_SWITCH(tensors[0].dtype(), IdType,
                    { return Unique<IdType>(total_tensor); });
  return std::make_tuple(torch::Tensor(), torch::Tensor(), torch::Tensor());
}

}  // namespace bifeat